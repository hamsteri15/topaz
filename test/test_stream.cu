#define CATCH_CONFIG_ENABLE_BENCHMARKING
#define CATCH_CONFIG_MAIN // This tells the catch header to generate a main
#include "catch.hpp"

#include "all.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_malloc_allocator.h>
#include <thrust/execution_policy.h>
#include <thrust/async/copy.h>
#include <thrust/async/transform.h>

template<class T>
using vector_t = thrust::device_vector<T>;

template<class T>
using NVec_t = topaz::NumericArray<T, thrust::device_malloc_allocator<T>>;

template <class Policy, class Range1_t, class Range2_t>
void copy(Policy p, const Range1_t& src, Range2_t& dst) {
    thrust::copy(p, src.begin(), src.end(), dst.begin());

}

std::vector<hipStream_t> create_streams(size_t count) {

    std::vector<hipStream_t> streams(count);
    for (auto& stream : streams){
        hipStreamCreate(&stream);
    }
    return streams;
}

void destroy_streams(std::vector<hipStream_t>& streams){

    for (auto& stream : streams){
        hipStreamDestroy(stream);
    }
}

template<class Vector_t>
auto arithmetic1(const Vector_t& v1, const Vector_t& v2, const Vector_t& v3){
    using T = typename Vector_t::value_type;
    return v1 * v2 + T(43) / v1 * v3 - v1 - T(32);
}

struct NoOp{

    CUDA_HOSTDEV double operator()(const double& d) {return d;}

};
TEST_CASE("asd"){


    size_t n_elements = 2000;
    size_t n_kernels = 400;

    using element_t = double;
    std::vector<NVec_t<element_t>> results(n_kernels, NVec_t<element_t>(n_elements, 0));
    NVec_t<element_t> v1(n_elements, 1);
    NVec_t<element_t> v2(n_elements, 2);
    NVec_t<element_t> v3(n_elements, 3);

    std::vector<thrust::host_vector<element_t>> results_host(n_kernels, std::vector<element_t>(n_elements));

    BENCHMARK("Sequential kernel evaluation"){

        for (size_t i = 0; i < n_kernels; ++i){
            auto kernel = arithmetic1(v1, v2, v3);
            topaz::copy(kernel, results[i]);
        }
        for (size_t i = 0; i < n_kernels; ++i){
            topaz::copy(results[i], results_host[i]);
        }
        return results_host;
    };

    BENCHMARK("Streamed kernel evaluation"){

        size_t n_streams = n_kernels;
        auto streams = create_streams(n_streams);


        for (size_t i = 0; i < n_kernels; ++i){
            auto kernel = arithmetic1(v1, v2, v3);
            auto policy = thrust::cuda::par.on(streams[i]);
            copy(policy, kernel, results[i]);
        }

        for (size_t i = 0; i < n_kernels; ++i){
            hipStreamSynchronize(streams[i]);
            topaz::copy(results[i], results_host[i]);
        }

        destroy_streams(streams);
        return results_host;
    };

    BENCHMARK("async copy"){


        std::vector<thrust::device_event> events;

        for (size_t i = 0; i < n_kernels; ++i){

            auto kernel = arithmetic1(v1, v2, v3);
            events.push_back(thrust::async::transform(
                kernel.begin(), kernel.end(), results[i].begin(), NoOp{}
            ));
            //events.push_back(event);
        }

        for (size_t i = 0; i < n_kernels; ++i){
            /*auto e = thrust::async::copy
            (
                thrust::host.after(events[i]),
                results[i].begin(), results[i].end(), results_host[i].begin()
            );
            e.wait();*/
            events[i].wait();
            topaz::copy(results[i], results_host[i]);
        }


        return results_host;
    };



}